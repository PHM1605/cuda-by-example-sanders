#include "../common/book.h"

int main(void) {
    hipDeviceProp_t prop;
    int dev;
    HANDLE_ERROR( hipGetDevice(&dev) );
    printf("ID of current Cuda device: %d\n", dev);

    // set config for which cuda device we want to pick out
    memset(&prop, 0, sizeof(hipDeviceProp_t));
    prop.major = 1;
    prop.minor = 3;

    // set to the collected device
    HANDLE_ERROR( hipChooseDevice(&dev, &prop) );
    printf("ID of CUDA device closest to revision 1.3: %d\n", dev);
    

    return 0;
}